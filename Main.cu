
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

__global__ void add(int n, float *x, float *y, float *z, float *o)
{

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (unsigned long i = 0; i < n; i++)
    {
        //std::cout << "iter " << i << std::endl;
        x[i] = (float)(536.0f / 32.23f);

        y[i] = (float)(653.1f / 24.354f);
    }

    for (int i = index; i < n; i += stride)
    {

        y[i] = x[i] + y[i];
        x[i] = y[i] * 0.5f;
    }
    for (int i = index; i < n; i += stride)
    {

        y[i] = (x[i] + (y[i] * 1.3f)) / (sqrt(y[i]) * cbrt(x[i] * 0.5f) / 3);
        x[i] = y[i] * 0.5f;
    }
}

int main(void)
{
    auto t1 = std::chrono::high_resolution_clock::now();

    std::cout << "Running 1" << std::endl;
    int n = 1 << 30; //1m

    float *x;
    float *y;
    float *z;
    float *o;
    //need to alloc accessible memory

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));
    hipMallocManaged(&z, n * sizeof(float));
    hipMallocManaged(&o, n * sizeof(float));

    //  std::cout << "Running 2 | Size: " << sizeof(x) << std::endl;

    //  std::cout << "Running 3" << std::endl;
    add<<<20, 1024>>>(n, x, y, z, o);

    add<<<20, 1024>>>(n, y, x, z, o);
    add<<<20, 1024>>>(n, x, z, o, y);
  
    //  for(int w = 0; w < n; w++){
    //       std::cout << y[w] << std::endl;;
    //   }

    //  std::cout << "Running 4" << std::endl;

    //CPU wait until task done
    hipDeviceSynchronize();
    //must call sync before freeing memory
    hipFree(x);
    hipFree(y);
    hipFree(o);
    hipFree(z);

    auto t2 = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

    std::cout << "Time taken: " << float(duration / 1000.00f) << "ms";
    return 0;
}