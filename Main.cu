
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

__global__ void add(int n, float *x, float *y)
{

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (unsigned long i = 0; i < n; i++)
    {
        //std::cout << "iter " << i << std::endl;
      
            x[i] = (float)(536.0f / 32.23f);

            y[i] = (float)(653.1f / 24.354f);
        
    }

    for (int i = index; i < n; i += stride)
    {
      
            y[i] = x[i] + y[i];
            x[i] = y[i] * 0.5f;
        
    }
    for (int i = index; i < n; i += stride)
    {
        
            y[i] = (x[i] + (y[i] * 1.3f)) / (sqrt(y[i]) * cbrt(x[i] * 0.5f) / 3);
            x[i] = y[i] * 0.5f;
        
    }
}

int main(void)
{
  //  auto t1 = std::chrono::high_resolution_clock::now();
    //for maximum GPU eating
    //  for (int loopsNum = 0; loopsNum < 5000; loopsNum += 1)
    //  {

  //  std::cout << "Running 1" << std::endl;
    int n = (1000000); //

    float *x;
    float *y;

    //need to alloc accessible memory

    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));
 

    //  std::cout << "Running 2 | Size: " << sizeof(x) << std::endl;

    //  std::cout << "Running 3" << std::endl;
    add<<<6, 128>>>(n,x,y);

    //  for(int w = 0; w < n; w++){
    //       std::cout << y[w] << std::endl;;
    //   }

    //  std::cout << "Running 4" << std::endl;

    //CPU wait until task done
    hipDeviceSynchronize();
    //must call sync before freeing memory
    hipFree(x);
    hipFree(y);

    // }
    //auto t2 = std::chrono::high_resolution_clock::now();

   // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

  //  std::cout << "Time taken: " << float(duration / 1000.00f) << "ms";
    return 0;
}