
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
__global__ void init(int length, float *a, float *b)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < length; i += stride)
    {
        a[i] = 2.3423;
        b[i] = 1.4214;
    }
}

__global__ void mMultiply(int length, float *a, float *b, float *output)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < length; i += stride)
    {
        output[i] = (a[i] * b[i]);
    }
    for (int i = index; i < length; i += stride)
    {
        output[i] = (output[i] * a[i]);
    }
    for (int i = index; i < length; i += stride)
    {
        output[i] = (output[i] * b[i]);
    }
}

int main(void)
{
    std::cout << "Starting CUDA example... Array 10m entries" << std::endl;
    std::cout << "Creating array" << std::endl;

    int number = 10000000;

    float *testA;
    float *testB;
    float *outputC;

    hipMallocManaged(&testA, number * sizeof(float));
    hipMallocManaged(&testB, number * sizeof(float));
    hipMallocManaged(&outputC, number * sizeof(float));
    /*
     for (int i = 0; i < number; i++)
    {
        testA[i] = 2.3423;
        testB[i] = 1.4214;
    }
    
*/

//    init<<<9766, 1024>>>(number, testA, testB);
    init<<<9766, 1024>>>(number, testA, testB);
    hipDeviceSynchronize();

    std::cout << "Completed array creation." << std::endl;
    std::cout << "Calculating..." << std::endl;

    auto time1 = std::chrono::high_resolution_clock::now();

    mMultiply<<<9766, 1024>>>(number, testA, testB, outputC);

    hipDeviceSynchronize();

    auto time2 = std::chrono::high_resolution_clock::now();

    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count();

    std::cout << "[CUDA] Time taken: " << (dur / 1000.00) << "ms";

    hipFree(testA);
    hipFree(testB);
    hipFree(outputC);
    return 0;
}