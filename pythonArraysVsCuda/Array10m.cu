
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

__global__ void mMultiply(int length, float *a, float *b, float *output)
{
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < length; i += stride)
    {
        output[i] = (a[i] * b[i]);
    }
    for (int i = index; i < length; i += stride)
    {
        output[i] = (output[i] * a[i]);
    }
    for (int i = index; i < length; i += stride)
    {
        output[i] = (output[i] * b[i]);
    }
}

int main(void)
{
    std::cout << "Starting CUDA example... Array 10m entries" << std::endl;
    std::cout << "Creating array" << std::endl;

    float *testA;
    float *testB;
    float *outputC;

    hipMallocManaged(&testA, 10000000 * sizeof(float));
    hipMallocManaged(&testB, 10000000 * sizeof(float));
    hipMallocManaged(&outputC, 10000000 * sizeof(float));

    std::cout << "Completed array creation." << std::endl;
    std::cout << "Calculating..." << std::endl;

    auto time1 = std::chrono::high_resolution_clock::now();

    mMultiply<<<5, 1024>>>(10000000, testA, testB, outputC);

    hipDeviceSynchronize();

    auto time2 = std::chrono::high_resolution_clock::now();

    auto dur = std::chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count();

    std::cout << "[CUDA] Time taken: " << (dur / 1000.00) << "ms";
  

    hipFree(testA);
    hipFree(testB);
    hipFree(outputC);
    return 0;
}